#include "hip/hip_runtime.h"

#include "../include/evolution.h"

void evolve_2d(Wave &wave, Op &opr,
               hipfftDoubleComplex *gpuParSum, int numSteps, Cuda &cupar,
               unsigned int gstate, Grid &par, 
               std::string buffer){

    // Re-establishing variables from parsed Grid class
    std::string data_dir = par.sval("data_dir");
    double omega = par.dval("omega");
    double angle_sweep = par.dval("angle_sweep");
    double gdt = par.dval("gdt");
    double dt = par.dval("dt");
    double omegaX = par.dval("omegaX");
    double omegaY = par.dval("omegaY");
    double omegaZ = par.dval("omegaZ");
    double mass = par.dval("mass");
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double interaction = par.dval("interaction");
    double laser_power = par.dval("laser_power");
    double gDenConst = par.dval("gDenConst");
    double DX = par.dval("DX");
    double mask_2d = par.dval("mask_2d");
    double *x = par.dsval("x");
    double *y = par.dsval("y");
    double *V = opr.dsval("V");
    double *V_opt = opr.dsval("V_opt");
    double *Phi = wave.dsval("Phi");
    double *gpu1dpAx = opr.dsval("pAx_gpu");
    double *gpu1dpAy = opr.dsval("pAy_gpu");
    double *Phi_gpu = wave.dsval("Phi_gpu");
    int kick_it = par.ival("kick_it");
    bool write_it = par.bval("write_it");
    bool graph = par.bval("graph");
    int N = par.ival("atoms");
    int printSteps = par.ival("printSteps");
    bool nonlin = par.bval("gpe");
    bool lz = par.bval("corotating");
    bool ramp = par.bval("ramp");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int gridSize = xDim * yDim;
    int kill_idx = par.ival("kill_idx");
    hipfftDoubleComplex *EV = opr.cufftDoubleComplexval("EV");
    hipfftDoubleComplex *wfc = wave.cufftDoubleComplexval("wfc");
    hipfftDoubleComplex *EV_opt = opr.cufftDoubleComplexval("EV_opt");
    hipfftDoubleComplex *gpuWfc = wave.cufftDoubleComplexval("wfc_gpu");
    hipfftDoubleComplex *K_gpu =
        opr.cufftDoubleComplexval("K_gpu");
    hipfftDoubleComplex *V_gpu =
        opr.cufftDoubleComplexval("V_gpu");

    std::cout << x[0] << '\t' << EV[0].x << '\t' << wfc[0].x << '\t'
              << EV_opt[0].x << '\t' << '\n';

    // getting data from Cuda class
    hipfftResult result = cupar.cufftResultval("result");
    hipfftHandle plan_1d = cupar.cufftHandleval("plan_1d");
    hipfftHandle plan_2d = cupar.cufftHandleval("plan_2d");
    hipfftHandle plan_other2d = cupar.cufftHandleval("plan_other2d");

    dim3 threads = cupar.dim3val("threads");
    dim3 grid = cupar.dim3val("grid");

    // Because no two operations are created equally. 
    // Multiplication is faster than divisions.
    double renorm_factor_2d=1.0/pow(gridSize,0.5);
    double renorm_factor_1d=1.0/pow(xDim,0.5);

    // outputting a bunch of variables just to check thigs out...
    std::cout << omega << '\t' << angle_sweep << '\t' << gdt << '\t'
              << dt << '\t' << omegaX << '\t' << omegaY << '\t' 
              << mass << '\t' << dx << '\t' << dy << '\t' << interaction << '\t'
              << laser_power << '\t' << N << '\t' << xDim << '\t' 
              << yDim << '\n';


    clock_t begin, end;
    double time_spent;
    double Dt;
    if(gstate==0){
        Dt = gdt;
        printf("Timestep for groundstate solver set as: %E\n",Dt);
    }
    else{
        Dt = dt;
        printf("Timestep for evolution set as: %E\n",Dt);
    }
    begin = clock();
    double omega_0=omega*omegaX;

    // ** ############################################################## ** //
    // **         HERE BE DRAGONS OF THE MOST DANGEROUS KIND!            ** //
    // ** ############################################################## ** //

    // Double buffering and will attempt to thread free and calloc operations to
    // hide time penalty. Or may not bother.
    int num_vortices[2] = {0,0};

    // binary matrix of size xDim*yDim, 
    // 1 for vortex at specified index, 0 otherwise
    int* vortexLocation;
    int* olMaxLocation = (int*) calloc(xDim*yDim,sizeof(int));

    struct Vtx::Vortex central_vortex; //vortex closest to the central position

    // Angle of vortex lattice. Add to optical lattice for alignment.
    double vort_angle;

    // array of vortex coordinates from vortexLocation 1's
    struct Vtx::Vortex *vortCoords = NULL;

    //Previous array of vortex coordinates from vortexLocation 1's
    struct Vtx::Vortex *vortCoordsP = NULL;

    LatticeGraph::Lattice lattice; //Vortex lattice graph.
    double* adjMat;
    
    double vortOLSigma=0.0;
    double sepAvg = 0.0;
    
    int num_kick = 0;
    double t_kick = (2*PI/omega_0)/(6*Dt);

    //std::cout << "numSteps is: " << numSteps << '\n';
    // Iterating through all of the steps in either g or esteps.
    for(int i=0; i < numSteps; ++i){
        if ( ramp ){
            //Adjusts omega for the appropriate trap frequency.
            omega_0=omegaX*((omega-0.39)*((double)i/(double)(numSteps)) + 0.39);
        }

        // Print-out at pre-determined rate.
        // Vortex & wfc analysis performed here also.
        if(i % printSteps == 0) { 
            // If the unit_test flag is on, we need a special case
            printf("Step: %d    Omega: %lf\n", i, omega_0 / omegaX);
            hipMemcpy(wfc, gpuWfc, sizeof(hipfftDoubleComplex) * xDim * yDim, 
                       hipMemcpyDeviceToHost);

            // Printing out time of iteration
            end = clock();
            time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
            printf("Time spent: %lf\n", time_spent);
            std::string fileName = "";
            printf("ramp=%d        gstate=%d    rg=%d        \n", 
                   ramp, gstate, ramp | (gstate << 1));
            switch (ramp | (gstate << 1)) {
                case 0: //Groundstate solver, constant Omega value.
                    std::cout << "we are in case 0" << '\n';
                    fileName = "wfc_0_const";
                    break;
                case 1: //Groundstate solver, ramped Omega value.
                    std::cout << "we are in state 1" << '\n';
                    fileName = "wfc_0_ramp";
                    break;
                case 2: //Real-time evolution, constant Omega value.
                    std::cout << "we are in case 2" << '\n';
                    fileName = "wfc_ev";
                    vortexLocation = (int *) calloc(xDim * yDim, sizeof(int));
                    num_vortices[0] = Tracker::findVortex(vortexLocation, wfc,
                                                          mask_2d, xDim, x, i);

                    // If initial step, locate vortices, least-squares to find
                    // exact centre, calculate lattice angle, generate optical 
                    // lattice.
                    if (i == 0) {
                        vortCoords = (struct Vtx::Vortex *) malloc(
                                sizeof(struct Vtx::Vortex) * 
                                (2 * num_vortices[0]));
                        vortCoordsP = (struct Vtx::Vortex *) malloc(
                                sizeof(struct Vtx::Vortex) * 
                                (2 * num_vortices[0]));
                        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
                        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
                        central_vortex = Tracker::vortCentre(vortCoords, 
                                                             num_vortices[0], 
                                                             xDim);
                        vort_angle = Tracker::vortAngle(vortCoords, 
                                                        central_vortex, 
                                                        num_vortices[0]);
                        par.store("Vort_angle", vort_angle);
                        optLatSetup(central_vortex, V, vortCoords, 
                                    num_vortices[0], 
                                    vort_angle + PI * angle_sweep / 180.0,
                                    laser_power * HBAR * sqrt(omegaX * omegaY),
                                    V_opt, x, y, par, opr);
                        //V = opr.dsval("V");
                        //V_opt = opr.dsval("V_opt");
                        //EV_opt = opr.cufftDoubleComplexval("EV_opt");
                        sepAvg = Tracker::vortSepAvg(vortCoords, central_vortex,
                                                     num_vortices[0]);
                        if (kick_it == 2) {
                            printf("Kicked it 1\n");
                            hipMemcpy(V_gpu, EV_opt, 
                                       sizeof(hipfftDoubleComplex) * xDim * yDim,
                                       hipMemcpyHostToDevice);
                        }
                        FileIO::writeOutDouble(buffer, data_dir + "V_opt_1",
                                               V_opt, xDim * yDim, 0);
                        FileIO::writeOut(buffer, data_dir + "EV_opt_1", EV_opt, 
                                         xDim * yDim, 0);
                        par.store("Central_vort_x", 
                                  (double) central_vortex.coords.x);
                        par.store("Central_vort_y", 
                                  (double) central_vortex.coords.y);
                        par.store("Central_vort_winding", 
                                  (double) central_vortex.wind);
                        par.store("Num_vort", (double) num_vortices[0]);
                        //std::cout << "writing to file in conditional" << '\n';
                        FileIO::writeOutParam(buffer, par, 
                                              data_dir + "Params.dat");
                    }
                    else if (num_vortices[0] > num_vortices[1]) {
                        printf("Number of vortices increased from %d to %d\n", 
                               num_vortices[1], num_vortices[0]);
                        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
                        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
                    }
                    // if num_vortices[1] < num_vortices[0] ... Fewer vortices
                    else {
                        Tracker::vortPos(vortexLocation, vortCoords, xDim, wfc);
                        Tracker::lsFit(vortCoords, wfc, num_vortices[0], xDim);
                        Tracker::vortArrange(vortCoords, vortCoordsP, 
                                             num_vortices[0]);
                    }

                    // The following will be modified and moved into a new 
                    // library that works closely with GPUE
                    if (graph) {

                        for (int ii = 0; ii < num_vortices[0]; ++ii) {
                            std::shared_ptr<LatticeGraph::Node> 
                                n(new LatticeGraph::Node(vortCoords[ii]));
                            lattice.addVortex(std::move(n));
                        }
                        unsigned int *uids = (unsigned int *) malloc(
                                sizeof(unsigned int) *
                                lattice.getVortices().size());
                        for (size_t a=0; a < lattice.getVortices().size(); ++a){
                            uids[a] = lattice.getVortexIdx(a)->getUid();
                        }
                        if(i==0) {
                            //Lambda for vortex annihilation/creation.
                            auto killIt=[&](int idx, int winding, 
                                            double delta_x) {
                                WFC::phaseWinding(Phi, 1, x, y, dx, dy,
                                    lattice.getVortexUid(idx)->
                                    getData().coordsD.x 
                                    +cos(angle_sweep + vort_angle)*delta_x,
                                    lattice.getVortexUid(idx)->
                                    getData().coordsD.y
                                    +sin(angle_sweep + vort_angle)*delta_x,
                                    xDim);
                                hipMemcpy(Phi_gpu, Phi, 
                                           sizeof(double) * xDim * yDim, 
                                           hipMemcpyHostToDevice);
                                cMultPhi <<<grid, threads>>> (gpuWfc, Phi_gpu, 
                                                              gpuWfc);
                            };
                            if (kill_idx > 0){
                                killIt(kill_idx, 1, DX);
                            }

                        }
                        lattice.createEdges(1.5 * 2e-5 / dx);
                        adjMat = (double *)calloc(lattice.getVortices().size() *
                                                  lattice.getVortices().size(),
                                                   sizeof(double));
                        lattice.genAdjMat(adjMat);
                        FileIO::writeOutAdjMat(buffer, data_dir + "graph", 
                                               adjMat, uids, 
                                               lattice.getVortices().size(), i);
                        free(adjMat);
                        free(uids);
                        lattice.getVortices().clear();
                        lattice.getEdges().clear();
                        //exit(0);
                    }

                    FileIO::writeOutVortex(buffer, data_dir + "vort_arr",
                                           vortCoords, num_vortices[0], i);
                    printf("Located %d vortices\n", num_vortices[0]);
                    printf("Sigma=%e\n", vortOLSigma);
                    free(vortexLocation);
                    num_vortices[1] = num_vortices[0];
                    memcpy(vortCoordsP, vortCoords, 
                           sizeof(int2) * num_vortices[0]);
                    //exit(1);
                    //std::cout << "finished case 2" << '\n';
                    break;

                case 3:
                    fileName = "wfc_ev_ramp";
                    break;
                default:
                    break;
            }

            //std::cout << "writing" << '\n';
            if (write_it) {
                FileIO::writeOut(buffer, data_dir + fileName, 
                                 wfc, xDim * yDim, i);
            }
            //std::cout << "written" << '\n';
            //printf("Energy[t@%d]=%E\n",i,energy_angmom(V_gpu, 
            //       K_gpu, dx, dy, gpuWfc,gstate));
        }

        // No longer writing out

        // ** ########################################################## ** //
        // **                     More F'n' Dragons!                     ** //
        // ** ########################################################## ** //

        // If not already kicked at this time step more than 6 times... kick it!
        if(i%((int)t_kick+1) == 0 && num_kick<=6 && gstate==1 && kick_it == 1 ){
            hipMemcpy(V_gpu, EV_opt, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
            ++num_kick;
        }
        // ** ########################################################## ** //

        // U_r(dt/2)*wfc
        if(nonlin == 1){
            //std::cout << Dt << '\t' << mass << '\t' << omegaZ << '\t' 
            //          << gstate << '\t' << N*interaction << '\n';
            cMultDensity<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc,0.5*Dt,
                                           mass,gstate,interaction*gDenConst);
        }
        else {
            cMult<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc);
        }
                
        // U_p(dt)*fft2(wfc)
        result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_FORWARD);

        // Normalise
        scalarMult<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
        cMult<<<grid,threads>>>(K_gpu,gpuWfc,gpuWfc);
        result = hipfftExecZ2Z(plan_2d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);

        // Normalise
        scalarMult<<<grid,threads>>>(gpuWfc,renorm_factor_2d,gpuWfc);
        
        // U_r(dt/2)*wfc
        if(nonlin == 1){
            cMultDensity<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc,Dt*0.5,
                                           mass,gstate,interaction*gDenConst);
        }
        else {
            cMult<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc);
        }

        // If first timestep and kick_it >= 1, kick.
        // Also kick if not kicked enough
        if( (i % (int)(t_kick+1) == 0 && num_kick<=6 && gstate==1) || 
            (kick_it >= 1 && i==0) ){
            hipMemcpy(V_gpu, EV, sizeof(hipfftDoubleComplex)*xDim*yDim, 
                       hipMemcpyHostToDevice);
            printf("Got here: Cuda memcpy EV into GPU\n");
        }
        // Angular momentum pAy-pAx (if engaged)  //
        if(lz == 1){
            // Multiplying by ramping factor if necessary
            // Note: using scalarPow to do the scaling inside of the exp
            if (ramp ){
                scalarPow<<<grid,threads>>>((hipfftDoubleComplex*) gpu1dpAy, 
                                            omega_0/(omega * omegaY),
                                            (hipfftDoubleComplex*) gpu1dpAy);
                scalarPow<<<grid,threads>>>((hipfftDoubleComplex*) gpu1dpAx, 
                                            omega_0/(omega * omegaX),
                                            (hipfftDoubleComplex*) gpu1dpAx);
            }
            switch(i%2 | (gstate<<1)){
                case 0: //Groundstate solver, even step

                    // 1d forward / mult by Ay
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);


                    // 1D FFT to wfc_pAx
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_FORWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAx, gpuWfc);
    
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);
                    break; 
                
                case 1:    //Groundstate solver, odd step
                    // 1D FFT to wfc_pAx
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_FORWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAx, gpuWfc);
    
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);

                    // wfc_pAy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);
                    break; 
                
                case 2: //Real time evolution, even step
                    //std::cout << "RT solver even." << '\n';

                    // wfc_pAy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                
                    // 1D to wfc_pAx
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_FORWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAx, gpuWfc);

                    // wfc_pAy
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_BACKWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);

                    break;
                
                case 3:    //Real time evolution, odd step
                    //std::cout << "RT solver odd." << '\n';

                    // 1D inverse to wfc_pAx
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAx, gpuWfc);

                    // wfc_pAy
                    result = hipfftExecZ2Z(plan_other2d,gpuWfc,gpuWfc,
                                          HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);

                    // wfc_pAy
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAy, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    break;
            
            }
        }
    
        if(gstate==0){
            parSum(gpuWfc, gpuParSum, par, cupar);
        }
    }

    // std::cout << "finished evolution" << '\n';
    // Storing wavefunctions for later
    //std::cout << gpuWfc[0].x << '\t' << gpuWfc[0].y << '\n';
    wave.store("wfc", wfc);
    wave.store("wfc_gpu", gpuWfc);

/*
    par.store("omega", omega);
    par.store("angle_sweep", angle_sweep);
    par.store("gdt", gdt);
    par.store("dt", dt);
    par.store("omegaX", omegaX);
    par.store("omegaY", omegaY);
    par.store("omegaZ", omegaZ);
    par.store("mass", mass);
    par.store("dx", dx);
    par.store("dy", dy);
    par.store("interaction", interaction);
    par.store("laser_power", laser_power);
    par.store("x", x);
    par.store("y", y);
    opr.store("V", V);
    opr.store("V_opt", V_opt);
    wave.store("Phi", Phi);
    opr.store("pAx_gpu", gpu1dpAx);
    opr.store("pAy_gpu", gpu1dpAy);
    wave.store("Phi_gpu", Phi_gpu);
    opr.store("EV", EV);
    //opr.store("V_gpu", V_gpu);
    //opr.store("K_gpu", K_gpu);
    opr.store("EV_opt", EV_opt);

    // getting data from Cuda class
    cupar.store("result", result);
    cupar.store("plan_1d", plan_1d);
    cupar.store("plan_2d", plan_2d);
    cupar.store("grid", grid);
*/

}

/*----------------------------------------------------------------------------//
* 3D
* Notes: In this case, we need to think about how to do the vortex tracking
*        Kicking will also be hard to do... Though not impossible, I suppose.
*-----------------------------------------------------------------------------*/

void evolve_3d(Wave &wave, Op &opr,
               hipfftDoubleComplex *gpuParSum, int numSteps, Cuda &cupar,
               unsigned int gstate, Grid &par, 
               std::string buffer){

    // Re-establishing variables from parsed Grid class
    std::string data_dir = par.sval("data_dir");
    double omega = par.dval("omega");
    double angle_sweep = par.dval("angle_sweep");
    double gdt = par.dval("gdt");
    double dt = par.dval("dt");
    double omegaX = par.dval("omegaX");
    double omegaY = par.dval("omegaY");
    double omegaZ = par.dval("omegaZ");
    double mass = par.dval("mass");
    double dx = par.dval("dx");
    double dy = par.dval("dy");
    double dz = par.dval("dz");
    double interaction = par.dval("interaction");
    double laser_power = par.dval("laser_power");
    double gDenConst = par.dval("gDenConst");
    double *x = par.dsval("x");
    double *y = par.dsval("y");
    double *z = par.dsval("z");
    double *V = opr.dsval("V");
    double *V_opt = opr.dsval("V_opt");
    double *Phi = wave.dsval("Phi");
    double *gpu1dpAx = opr.dsval("pAx_gpu");
    double *gpu1dpAy = opr.dsval("pAy_gpu");
    double *gpu1dpAz = opr.dsval("pAz_gpu");
    double *Phi_gpu = wave.dsval("Phi_gpu");
    bool write_it = par.bval("write_it");
    bool graph = par.bval("graph");
    int N = par.ival("atoms");
    int printSteps = par.ival("printSteps");
    bool nonlin = par.bval("gpe");
    bool lz = par.bval("corotating");
    std::cout << "COROTATING IS: " << lz << '\n';
    bool ramp = par.bval("ramp");
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = par.ival("zDim");
    int gridSize = xDim * yDim * zDim;
    hipfftDoubleComplex *EV = opr.cufftDoubleComplexval("EV");
    hipfftDoubleComplex *wfc = wave.cufftDoubleComplexval("wfc");
    hipfftDoubleComplex *EV_opt = opr.cufftDoubleComplexval("EV_opt");
    hipfftDoubleComplex *gpuWfc = wave.cufftDoubleComplexval("wfc_gpu");
    hipfftDoubleComplex *K_gpu =
        opr.cufftDoubleComplexval("K_gpu");
    hipfftDoubleComplex *V_gpu =
        opr.cufftDoubleComplexval("V_gpu");

    std::cout << x[0] << '\t' << EV[0].x << '\t' << wfc[0].x << '\t'
              << EV_opt[0].x << '\t' << '\n';

    // getting data from Cuda class
    hipfftResult result = cupar.cufftResultval("result");
    hipfftHandle plan_1d = cupar.cufftHandleval("plan_1d");
    hipfftHandle plan_3d = cupar.cufftHandleval("plan_3d");
    hipfftHandle plan_dim2 = cupar.cufftHandleval("plan_dim2");
    hipfftHandle plan_dim3 = cupar.cufftHandleval("plan_dim3");
    dim3 threads = cupar.dim3val("threads");
    dim3 grid = cupar.dim3val("grid");

    // Because no two operations are created equally. 
    // Multiplication is faster than divisions.
    double renorm_factor_3d=1.0/pow(gridSize,0.5);
    double renorm_factor_1d=1.0/pow(xDim,0.5);

    // outputting a bunch of variables just to check thigs out...
    std::cout << omega << '\t' << angle_sweep << '\t' << gdt << '\t'
              << dt << '\t' << omegaX << '\t' << omegaY << '\t' 
              << mass << '\t' << dx << '\t' << dy << '\t' << interaction << '\t'
              << laser_power << '\t' << N << '\t' << xDim << '\t' 
              << yDim << '\n';


    clock_t begin, end;
    double time_spent;
    double Dt;
    if(gstate==0){
        Dt = gdt;
        printf("Timestep for groundstate solver set as: %E\n",Dt);
    }
    else{
        Dt = dt;
        printf("Timestep for evolution set as: %E\n",Dt);
    }
    begin = clock();
    double omega_0=omega*omegaX;

    // ** ############################################################## ** //
    // **         HERE BE DRAGONS OF THE MOST DANGEROUS KIND!            ** //
    // ** ############################################################## ** //

    //std::cout << "numSteps is: " << numSteps << '\n';
    // Iterating through all of the steps in either g or esteps.
    for(int i=0; i < numSteps; ++i){
        if (ramp){
            //Adjusts omega for the appropriate trap frequency.
            omega_0=omegaX*((omega-0.39)*((double)i/(double)(numSteps)) + 0.39);
        }

        // Print-out at pre-determined rate.
        // Vortex & wfc analysis performed here also.
        if(i % printSteps == 0) { 
            // If the unit_test flag is on, we need a special case
            printf("Step: %d    Omega: %lf\n", i, omega_0 / omegaX);
            hipMemcpy(wfc, gpuWfc, sizeof(hipfftDoubleComplex)*xDim*yDim*zDim, 
                       hipMemcpyDeviceToHost);

            // Printing out time of iteration
            end = clock();
            time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
            printf("Time spent: %lf\n", time_spent);
            std::string fileName = "";
            printf("ramp=%d        gstate=%d    rg=%d        \n", 
                   ramp, gstate, ramp | (gstate << 1));
            switch (ramp | (gstate << 1)) {
                case 0: //Groundstate solver, constant Omega value.
                    std::cout << "we are in case 0" << '\n';
                    fileName = "wfc_0_const";
                    break;
                case 1: //Groundstate solver, ramped Omega value.
                    std::cout << "we are in state 1" << '\n';
                    fileName = "wfc_0_ramp";
                    break;
                case 2: //Real-time evolution, constant Omega value.
                    // Note: In the case of 3d, we need to think about
                    //       vortex tracking in a new way.
                    //       It may be as simple as splitting the problem into
                    //       2D elements and working from there, but let's 
                    //       look into it when we need it in the future.
                    std::cout << "we are in case 2" << '\n';
                    fileName = "wfc_ev";
                    break;

                case 3:
                    fileName = "wfc_ev_ramp";
                    break;
                default:
                    break;
            }

            //std::cout << "writing" << '\n';
            if (write_it) {
                FileIO::writeOut(buffer, data_dir + fileName, 
                                 wfc, xDim*yDim*zDim, i);
            }
            //std::cout << "written" << '\n';
            //printf("Energy[t@%d]=%E\n",i,energy_angmom(V_gpu, 
            //       K_gpu, dx, dy, gpuWfc,gstate));
        }

        // No longer writing out

        // ** ########################################################## ** //
        // **                     More F'n' Dragons!                     ** //
        // ** ########################################################## ** //

        // U_r(dt/2)*wfc
        if(nonlin == 1){
            //std::cout << Dt << '\t' << mass << '\t' << omegaZ << '\t' 
            //          << gstate << '\t' << N*interaction << '\n';
            cMultDensity<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc,0.5*Dt,
                                           mass,gstate,interaction*gDenConst);
        }
        else {
            cMult<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc);
        }
                
        // U_p(dt)*fft2(wfc)
        result = hipfftExecZ2Z(plan_3d,gpuWfc,gpuWfc,HIPFFT_FORWARD);

        // Normalise
        scalarMult<<<grid,threads>>>(gpuWfc,renorm_factor_3d,gpuWfc);
        cMult<<<grid,threads>>>(K_gpu,gpuWfc,gpuWfc);
        result = hipfftExecZ2Z(plan_3d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);

        // Normalise
        scalarMult<<<grid,threads>>>(gpuWfc,renorm_factor_3d,gpuWfc);
        
        // U_r(dt/2)*wfc
        if(nonlin == 1){
            cMultDensity<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc,Dt*0.5,
                                           mass,gstate,interaction*gDenConst);
        }
        else {
            cMult<<<grid,threads>>>(V_gpu,gpuWfc,gpuWfc);
        }

        // Angular momentum pAy-pAx (if engaged)  //
        if(lz == true){
            // Multiplying by ramping factor if necessary
            // Note: using scalarPow to do the scaling inside of the exp
            if (ramp){
                scalarPow<<<grid,threads>>>((hipfftDoubleComplex*) gpu1dpAy, 
                                            omega_0/(omega * omegaY),
                                            (hipfftDoubleComplex*) gpu1dpAy);
                scalarPow<<<grid,threads>>>((hipfftDoubleComplex*) gpu1dpAx, 
                                            omega_0/(omega * omegaX),
                                            (hipfftDoubleComplex*) gpu1dpAx);
                scalarPow<<<grid,threads>>>((hipfftDoubleComplex*) gpu1dpAz, 
                                            omega_0/(omega * omegaZ),
                                            (hipfftDoubleComplex*) gpu1dpAz);
            }
            int size = xDim*zDim;
            switch(gstate){
                case 0: //Groundstate solver, even step

                    // 1d forward / mult by Az
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAz, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);

                    // loop to multiply by Ay
                    for (int i = 0; i < yDim; i++){
                        //size = xDim * zDim;
                        result = hipfftExecZ2Z(plan_dim2,
                                 &gpuWfc[i*size],
                                 &gpuWfc[i*size],HIPFFT_FORWARD);
                    }

                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAy, gpuWfc);

                    for (int i = 0; i < yDim; i++){
                        //size = xDim * zDim;
                        result = hipfftExecZ2Z(plan_dim2,
                                 &gpuWfc[i*size],
                                 &gpuWfc[i*size],HIPFFT_BACKWARD);
                    }
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);

                    // 1D FFT to wfc_pAx
                    result = hipfftExecZ2Z(plan_dim3,gpuWfc,gpuWfc,
                                          HIPFFT_FORWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAx, gpuWfc);
    
                    result = hipfftExecZ2Z(plan_dim3,gpuWfc,gpuWfc,
                                          HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);

                    break; 
                
                case 1: //Real time evolution, even step
                    // 1d forward / mult by Az
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_FORWARD); 
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAz, gpuWfc);
                    result = hipfftExecZ2Z(plan_1d,gpuWfc,gpuWfc,HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);

                    // loop to multiply by Ay
                    for (int i = 0; i < yDim; i++){
                        result = hipfftExecZ2Z(plan_dim2,
                                 &gpuWfc[i*xDim*zDim],
                                 &gpuWfc[i*xDim*zDim],HIPFFT_FORWARD);
                    }

                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAy, gpuWfc);

                    for (int i = 0; i < yDim; i++){
                        result = hipfftExecZ2Z(plan_dim2,
                                 &gpuWfc[i*xDim*zDim],
                                 &gpuWfc[i*xDim*zDim],HIPFFT_BACKWARD);
                    }
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);


                    // 1D FFT to wfc_pAx
                    result = hipfftExecZ2Z(plan_dim3,gpuWfc,gpuWfc,
                                          HIPFFT_FORWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d,gpuWfc);
                    cMult<<<grid,threads>>>(gpuWfc, 
                        (hipfftDoubleComplex*) gpu1dpAx, gpuWfc);
    
                    result = hipfftExecZ2Z(plan_dim3,gpuWfc,gpuWfc,
                                          HIPFFT_BACKWARD);
                    scalarMult<<<grid,threads>>>(gpuWfc,
                                                 renorm_factor_1d, gpuWfc);

                    break;
                
            }
        }
    
        if(gstate==0){
            parSum(gpuWfc, gpuParSum, par, cupar);
        }
    }

    // std::cout << "finished evolution" << '\n';
    // Storing wavefunctions for later
    //std::cout << gpuWfc[0].x << '\t' << gpuWfc[0].y << '\n';
    wave.store("wfc", wfc);
    wave.store("wfc_gpu", gpuWfc);
/*

    par.store("omega", omega);
    par.store("angle_sweep", angle_sweep);
    par.store("gdt", gdt);
    par.store("dt", dt);
    par.store("omegaX", omegaX);
    par.store("omegaY", omegaY);
    par.store("omegaZ", omegaZ);
    par.store("mass", mass);
    par.store("dx", dx);
    par.store("dy", dy);
    par.store("interaction", interaction);
    par.store("laser_power", laser_power);
    par.store("x", x);
    par.store("y", y);
    opr.store("V", V);
    opr.store("V_opt", V_opt);
    wave.store("Phi", Phi);
    opr.store("pAx_gpu", gpu1dpAx);
    opr.store("pAy_gpu", gpu1dpAy);
    wave.store("Phi_gpu", Phi_gpu);
    opr.store("EV", EV);
    //opr.store("V_gpu", V_gpu);
    //opr.store("K_gpu", K_gpu);
    opr.store("EV_opt", EV_opt);

    // getting data from Cuda class
    cupar.store("result", result);
    cupar.store("plan_1d", plan_1d);
    cupar.store("plan_2d", plan_2d);
    cupar.store("grid", grid);

*/
}
